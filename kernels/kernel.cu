#include "hip/hip_runtime.h"
#include <pycuda-complex.hpp>
#include <stdint.h>

typedef pycuda::complex<double> cmplx;

extern "C" __global__ void compute(uint8_t *Iter, cmplx pos,
                                   double zoom, uint8_t iterations, 
                                   uint8_t length){
    uint i = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint j = (blockIdx.y * blockDim.y) + threadIdx.y;
    uint idx = i + j*length;
    Iter[idx] = 0;
    cmplx Z = cmplx(0.0, 0.0);
    cmplx C = pos + cmplx(i, j) * zoom;
    for (int i=0; i < iterations; i++){
        Z = Z * Z + C;
        double bounded_d = Z.real() * Z.real() 
                         + Z.imag() * Z.imag();
        bool bounded = bounded_d < 2.0;
        Iter[idx] += bounded;
    }
}