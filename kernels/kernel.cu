#include "hip/hip_runtime.h"
#include <pycuda-complex.hpp>
#include <stdint.h>

typedef pycuda::complex<double> cmplx;

extern "C" __global__ void compute(cmplx *Z, cmplx *C, uint8_t *Iter,
                                   uint16_t height){
    uint i = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint j = (blockIdx.y * blockDim.y) + threadIdx.y;
    int idx = i*height + j;
    for (int i=0; i < 100; i++){
        Z[idx] = Z[idx] * Z[idx] + C[idx];
        double bounded_d = Z[idx].real() * Z[idx].real() 
                         + Z[idx].imag() * Z[idx].imag();
        bool bounded = bounded_d < 2.0;
        Iter[idx] += bounded;
    }
}